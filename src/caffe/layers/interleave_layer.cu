#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/interleave_layer.hpp"

#include <cstdio>

namespace caffe
{
  template <typename Dtype>
  __global__ void InterleaveForward( const int    nthreads
                                   , const Dtype* in_a
                                   , const Dtype* in_b
                                   , const Dtype* in_c 
                                   , const Dtype* in_d
                                   , const int    width
                                   , Dtype*       out_data )
  {
    CUDA_KERNEL_LOOP( index, nthreads )
    {
      int is_row_even = ( index / width ) % 2;
      int is_col_even = ( index % width ) % 2;
      if( !is_row_even )
      {
        
        if( !is_col_even )
        {
          //printf("in: %i, a_%i\n", index, ( index % width ) / 2 + index / width * 2 ) ;
          out_data[index] = in_a[ ( index % width ) / 2 + ( index / width ) * 2 ]; 
        }
	else
        {
          //printf("in: %i, b_%i\n", index, ( index % width ) / 2 + index / width * 2 ) ; 
          out_data[index] = in_b[ ( index % width ) / 2 + ( index / width ) * 2 ];
        }
      }
      else
      {
        if( !is_col_even )
        {        
          //printf("in: %i, c_%i\n", index, ( index % width ) / 2 + index / width * 2 - 2) ; 
          out_data[index] = in_c[ ( index % width ) / 2 + ( index / width ) * 2 - 2 ];
        }
        else
        {
          //printf("in: %i, d_%i\n", index, ( index % width ) / 2 + index / width * 2 - 2) ;
          out_data[index] = in_d[ ( index % width ) / 2 + ( index / width ) * 2 - 2]; 
        }
      }//if( !is_row_even )      
    }//CUDA_KERNEL_LOOP
  } 
 
  template <typename Dtype>
  void InterleaveLayer<Dtype>::Forward_gpu( const vector<Blob<Dtype>*>& bottom
                                          , const vector<Blob<Dtype>*>& top )
  {
    Dtype* top_data = top[0]->mutable_gpu_data();
    const int width = top[0]->shape(3);
    const int count = top[0]->count();

    //printf( "w: %i\n", width );

    const Dtype* bottom_data_a = bottom[0]->gpu_data();
    const Dtype* bottom_data_b = bottom[1]->gpu_data();
    const Dtype* bottom_data_c = bottom[2]->gpu_data();
    const Dtype* bottom_data_d = bottom[3]->gpu_data();

    InterleaveForward<Dtype>
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
        ( count, bottom_data_a, bottom_data_b, bottom_data_c, bottom_data_d, width, top_data );

    CUDA_POST_KERNEL_CHECK;
    
  }

  template <typename Dtype>
  __global__ void InterleaveBackward( const int nthreads
                                    , const Dtype* in_diff
                                    , const int width
                                    , Dtype* out_diff_a
                                    , Dtype* out_diff_b
                                    , Dtype* out_diff_c
                                    , Dtype* out_diff_d )
  {
    CUDA_KERNEL_LOOP( index, nthreads )
    {
      int is_row_even = ( index / width ) % 2;
      int is_col_even = ( index % width ) % 2;
      if( !is_row_even )
      {
        
        if( !is_col_even )
        {
          //printf("in: %i, a_%i\n", index, ( index % width ) / 2 + index / width * 2 ) ;
          out_diff_a[ ( index % width ) / 2 + ( index / width ) * 2 ] = in_diff[index]; 
        }
	else
        {
          //printf("in: %i, b_%i\n", index, ( index % width ) / 2 + index / width * 2 ) ; 
          out_diff_b[ ( index % width ) / 2 + ( index / width ) * 2 ] = in_diff[index];
        }
      }
      else
      {
        if( !is_col_even )
        {        
          //printf("in: %i, c_%i\n", index, ( index % width ) / 2 + index / width * 2 - 2) ; 
          out_diff_c[ ( index % width ) / 2 + ( index / width ) * 2 - 2 ] = in_diff[index];
        }
        else
        {
          //printf("in: %i, d_%i\n", index, ( index % width ) / 2 + index / width * 2 - 2) ;
          out_diff_d[ ( index % width ) / 2 + ( index / width ) * 2 - 2] = in_diff[index]; 
        }
      }//if
    }
  }

  template <typename Dtype>
  void InterleaveLayer<Dtype>::Backward_gpu( const vector<Blob<Dtype>*>& top
                                           , const vector<bool>& propagate_down
                                           , const vector<Blob<Dtype>*>& bottom )
  {
    //if( propagate_down[0] )
    //{
      const Dtype* top_diff = top[0]->gpu_diff();
      const int count	    = top[0]->count();
      const int width       = top[0]->shape(3);
  
      Dtype* bottom_diff_a = bottom[0]->mutable_gpu_diff();
      Dtype* bottom_diff_b = bottom[1]->mutable_gpu_diff();
      Dtype* bottom_diff_c = bottom[2]->mutable_gpu_diff();
      Dtype* bottom_diff_d = bottom[3]->mutable_gpu_diff(); 

      InterleaveBackward<Dtype>
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
        ( count, top_diff, width, bottom_diff_a, bottom_diff_b, bottom_diff_c, bottom_diff_d );

      CUDA_POST_KERNEL_CHECK; 
    //}
  }



















  INSTANTIATE_LAYER_GPU_FUNCS(InterleaveLayer);
  
} //namespace caffe
